#include "hip/hip_runtime.h"
/*
* Source code for this lab class is modifed from the book CUDA by Exmaple and provided by permission of NVIDIA Corporation
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include "hip/hip_runtime.h"
#include ""

#define IMAGE_DIM 2048
#define SPHERE_SIZE_SAMPLES 8
// Ex 2.3, Modify this value!
#define STARTING_SPHERES 16
#define MAX_SPHERES STARTING_SPHERES<<(SPHERE_SIZE_SAMPLES -1)

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF     2e10f

void output_image_file(uchar4* image);
void checkCUDAError(const char *msg);

struct Sphere {
	float   r, b, g;
	float   radius;
	float   x, y, z;
};

/* Device Code */

__device__ float sphere_intersect(Sphere *s, float ox, float oy, float *n) {
	float dx = ox - s->x;
	float dy = oy - s->y;
	float radius = s->radius;
	if (dx*dx + dy*dy < radius*radius) {
		float dz = sqrtf(radius*radius - dx*dx - dy*dy);
		*n = dz / sqrtf(radius * radius);
		return dz + s->z;
	}
	return -INF;
}

// Ex 2.1 (1/2), const __restrict__ notifies the compiler that the memory behind the named pointer
// will only be read via the name pointer (it is not aliased), and never written to,
// enabling it to optimise accesses with the read-only cache
// This does not guarantee that the compiler will decide to use the read-only cache
// __ldg() can be used to force memory accesses via the read-only cache, however this is limited to specific types
// See documentation: https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#ldg-function
__device__ float sphere_intersect_read_only(Sphere const* __restrict__ s, float ox, float oy, float *n) {
	float dx = ox - s->x;
	float dy = oy - s->y;
	float radius = s->radius;
	if (dx*dx + dy*dy < radius*radius) {
		float dz = sqrtf(radius*radius - dx*dx - dy*dy);
		*n = dz / sqrtf(radius * radius);
		return dz + s->z;
	}
	return -INF;
}

// Ex 2.2 (1/3), Constant memory is declared similar to device symbols, and will be allocated at compile time
__constant__ Sphere d_const_s[MAX_SPHERES];
__constant__ unsigned int d_sphere_count;

__global__ void ray_trace(uchar4 *image, Sphere *d_s) {
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	float   ox = (x - IMAGE_DIM / 2.0f);
	float   oy = (y - IMAGE_DIM / 2.0f);

	float   r = 0, g = 0, b = 0;
	float   maxz = -INF;
	for (int i = 0; i<d_sphere_count; i++) {
		Sphere *s = &d_s[i];
		float   n;
		float   t = sphere_intersect(s, ox, oy, &n);
		if (t > maxz) {
			float fscale = n;
			r = s->r * fscale;
			g = s->g * fscale;
			b = s->b * fscale;
			maxz = t;
		}
	}

	image[offset].x = (int)(r * 255);
	image[offset].y = (int)(g * 255);
	image[offset].z = (int)(b * 255);
	image[offset].w = 255;
}

// Ex 2.2 (2/3), Const cache accesses are most efficient when all threads in a warp are accessing the same constant cache value
// If threads in a warp, access different const cache locations simultaneously, then the accesses must be replayed
// once per unique location, potentially 32 times! This can be slower than basic global memory.
__global__ void ray_trace_const(uchar4 *image) {
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	float   ox = (x - IMAGE_DIM / 2.0f);
	float   oy = (y - IMAGE_DIM / 2.0f);

	float   r = 0, g = 0, b = 0;
	float   maxz = -INF;
	for (int i = 0; i<d_sphere_count; i++) {
		Sphere *s = &d_const_s[i];
		float   n;
		float   t = sphere_intersect(s, ox, oy, &n);
		if (t > maxz) {
			float fscale = n;
			r = s->r * fscale;
			g = s->g * fscale;
			b = s->b * fscale;
			maxz = t;
		}
	}

	image[offset].x = (int)(r * 255);
	image[offset].y = (int)(g * 255);
	image[offset].z = (int)(b * 255);
	image[offset].w = 255;
}
// Ex 2.1 (2/2), Part 2, see notes for part 1.
__global__ void ray_trace_read_only(uchar4 *image, Sphere const* __restrict__ d_s) {
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	float   ox = (x - IMAGE_DIM / 2.0f);
	float   oy = (y - IMAGE_DIM / 2.0f);

	float   r = 0, g = 0, b = 0;
	float   maxz = -INF;
	for (int i = 0; i<d_sphere_count; i++) {
		Sphere const* __restrict__ s = &d_s[i];
		float   n;
		float   t = sphere_intersect_read_only(s, ox, oy, &n);
		if (t > maxz) {
			float fscale = n;
			r = s->r * fscale;
			g = s->g * fscale;
			b = s->b * fscale;
			maxz = t;
		}
	}

	image[offset].x = (int)(r * 255);
	image[offset].y = (int)(g * 255);
	image[offset].z = (int)(b * 255);
	image[offset].w = 255;
}

/* Host code */

int main(void) {
	unsigned int image_size, spheres_size;
	uchar4 *d_image;
	uchar4 *h_image;
	hipEvent_t     start, stop;
	Sphere h_s[MAX_SPHERES];
	Sphere *d_s;
	float3 timing_data[SPHERE_SIZE_SAMPLES]; //timing data for SPHERE_SIZE_SAMPLES sphere counts where [0]=normal, [1]=read-only, [2]=const

	image_size = IMAGE_DIM*IMAGE_DIM*sizeof(uchar4);
	spheres_size = sizeof(Sphere)*MAX_SPHERES;

	// create timers
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate memory on the GPU for the output image
	hipMalloc((void**)&d_image, image_size);
	hipMalloc((void**)&d_s, spheres_size);
	checkCUDAError("CUDA malloc");

	// create some random spheres
	for (int i = 0; i<MAX_SPHERES; i++) {
		h_s[i].r = rnd(1.0f);
		h_s[i].g = rnd(1.0f);
		h_s[i].b = rnd(1.0f);
		h_s[i].x = rnd((float)IMAGE_DIM) - (IMAGE_DIM / 2.0f);
		h_s[i].y = rnd((float)IMAGE_DIM) - (IMAGE_DIM / 2.0f);
		h_s[i].z = rnd((float)IMAGE_DIM) - (IMAGE_DIM / 2.0f);
		h_s[i].radius = rnd(100.0f) + 20;
	}
	// Ex 2.2 (3/3), Copy data to constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(d_const_s), h_s, spheres_size);
	//copy to device memory
	hipMemcpy(d_s, h_s, spheres_size, hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy to device");

	//generate host image
	h_image = (uchar4*)malloc(image_size);

	//cuda layout
	dim3    blocksPerGrid(IMAGE_DIM / 16, IMAGE_DIM / 16);
	dim3    threadsPerBlock(16, 16);

	for (int i = 0; i < SPHERE_SIZE_SAMPLES; i++){
		unsigned int sphere_count = STARTING_SPHERES << i;
		printf("Executing code for sphere count %d\n", sphere_count);
		hipMemcpyToSymbol(HIP_SYMBOL(d_sphere_count), &sphere_count, sizeof(unsigned int));
		checkCUDAError("CUDA copy sphere count to device");

		// generate a image from the sphere data
		hipEventRecord(start, 0);
		ray_trace << <blocksPerGrid, threadsPerBlock >> >(d_image, d_s);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&timing_data[i].x, start, stop);
		checkCUDAError("kernel (normal)");

		// generate a image from the sphere data (using read only cache)
		hipEventRecord(start, 0);
		ray_trace_read_only << <blocksPerGrid, threadsPerBlock >> >(d_image, d_s);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&timing_data[i].y, start, stop);
		checkCUDAError("kernel (read-only)");

		// generate a image from the sphere data (using constant cache)
		hipEventRecord(start, 0);
		ray_trace_const << <blocksPerGrid, threadsPerBlock >> >(d_image);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&timing_data[i].z, start, stop);
		checkCUDAError("kernel (const)");
	}

	// copy the image back from the GPU for output to file
	hipMemcpy(h_image, d_image, image_size, hipMemcpyDeviceToHost);
	checkCUDAError("CUDA memcpy from device");

	//output timings
	printf("Timing Data Table\n Spheres | Normal | Read-only | Const\n");
	for (int i = 0; i < SPHERE_SIZE_SAMPLES; i++){
		int sphere_count = STARTING_SPHERES << i;
		printf(" %-7i | %-6.3f | %-9.3f | %.3f\n", sphere_count, timing_data[i].x, timing_data[i].y, timing_data[i].z);
	}

	// output image
	output_image_file(h_image);

	//cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(d_image);
	hipFree(d_s);
	free(h_image);

	return 0;
}

void output_image_file(uchar4* image)
{
	FILE *f; //output file handle

	//open the output file and write header info for PPM filetype
	f = fopen("output.ppm", "wb");
	if (f == NULL){
		fprintf(stderr, "Error opening 'output.ppm' output file\n");
		exit(1);
	}
	fprintf(f, "P6\n");
	fprintf(f, "# COM4521 Lab 05 Exercise02\n");
	fprintf(f, "%d %d\n%d\n", IMAGE_DIM, IMAGE_DIM, 255);
	for (int x = 0; x < IMAGE_DIM; x++){
		for (int y = 0; y < IMAGE_DIM; y++){
			int i = x + y*IMAGE_DIM;
			fwrite(&image[i], sizeof(unsigned char), 3, f); //only write rgb (ignoring a)
		}
	}

	fclose(f);
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
