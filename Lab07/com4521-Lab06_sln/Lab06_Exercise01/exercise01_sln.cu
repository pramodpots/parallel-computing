#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""


#define A_WIDTH 1024
#define A_HEIGHT 1024
#define B_WIDTH 1024
#define B_HEIGHT 1024
#define C_WIDTH B_WIDTH
#define C_HEIGHT A_HEIGHT

#define BLOCK_SIZE 8
#define NUM_SUBS (A_WIDTH / BLOCK_SIZE)

__device__ float d_A[A_HEIGHT][A_WIDTH];
__device__ float d_B[B_HEIGHT][B_WIDTH];
__device__ float d_C[C_HEIGHT][C_WIDTH];

float h_A[A_HEIGHT][A_WIDTH];
float h_B[B_HEIGHT][B_WIDTH];
float h_C[C_HEIGHT][C_WIDTH];
float h_C_ref[C_HEIGHT][C_WIDTH];

void checkCUDAError(const char *msg);
void matrixMulCPU(float A[A_HEIGHT][A_WIDTH], float B[B_HEIGHT][B_WIDTH], float C[C_HEIGHT][C_WIDTH]);
int matrixMulTest(float C[C_HEIGHT][C_WIDTH], float Cref[C_HEIGHT][C_WIDTH]);

__global__ void matrixMulCUDA()
{
    // Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int x = bx*BLOCK_SIZE + tx;
	int y = by*BLOCK_SIZE + ty;
    

	float Csub = 0;
	//iterate A_WIDTH (same as B_HEIGHT) to calculate the product
	for (int k = 0; k < A_WIDTH; k++){
		Csub += d_A[y][k] * d_B[k][x]; 
	}

	// Store the product value of C matrix
	d_C[y][x] = Csub;
}

__global__ void matrixMulCUDASharedMemory()
{
	__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	float Csub = 0;

	for (int i = 0; i < NUM_SUBS; i++){
		// Calculate indices of A and B matrix required to load the shared block of memory
		int a_x = (i*BLOCK_SIZE) + tx;
		int a_y = (by*BLOCK_SIZE) + ty;
		int b_x = (bx*BLOCK_SIZE) + tx;
		int b_y = (i*BLOCK_SIZE) + ty;

		As[ty][tx] = d_A[a_y][a_x];		// global memory load with stride of 1, SM bank with stride of 1 :-)
		Bs[ty][tx] = d_B[b_y][b_x];		// global memory load with stride of 1, SM bank with stride of 1 :-)

		// Sync to ensure sub matrix is fully loaded
		__syncthreads();

		// sum products of A and B sub matrices
		for (int k = 0; k < BLOCK_SIZE; ++k)
		{
			Csub += As[ty][k] * Bs[k][tx];		// conflict free loads regardless of BLOCK_SIZE due to bradcast read and stride of 1
		}

		// Sync to prevent run ahead (blocks loading new SM values before others have completed)
		__syncthreads();
	}

	// Store the product value of C matrix
	int c_x = (bx*BLOCK_SIZE) + tx;
	int c_y = (by*BLOCK_SIZE) + ty;
	d_C[c_y][c_x] = Csub;	// global memory load with stride of 1 :-)
}


int main(int argc, char **argv)

{
	unsigned int mem_size_A, mem_size_B, mem_size_C;
	unsigned int x, y, errors;
	float msec, occupancy;
	hipEvent_t start, stop;

	if (A_WIDTH != B_HEIGHT){
		printf("Error: A_HEIGHT and B_WIDTH do not match\n");
	}

	mem_size_A = sizeof(float)* A_WIDTH* A_HEIGHT;
	mem_size_B = sizeof(float)* B_WIDTH* B_HEIGHT;
	mem_size_C = sizeof(float)* C_WIDTH* C_HEIGHT;

	// Initialise A
	for (y = 0; y < A_HEIGHT; y++)
	for (x = 0; x <A_WIDTH; x++)
		h_A[y][x] = (float)rand() / RAND_MAX;
	// Initialise B
	for (y = 0; y < B_HEIGHT; y++)
	for (x = 0; x <B_WIDTH; x++)
		h_B[y][x] = (float)rand() / RAND_MAX;

	// copy host memory to device
	hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, mem_size_A);
	hipMemcpyToSymbol(HIP_SYMBOL(d_B), h_B, mem_size_B);
	checkCUDAError("CUDA memcpy");

	// Allocate CUDA events that we'll use for timing
	hipEventCreate(&start);
	hipEventCreate(&stop);
	checkCUDAError("CUDA event creation");

	// Setup execution parameters
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(C_WIDTH / BLOCK_SIZE, C_HEIGHT / BLOCK_SIZE);
	hipEventRecord(start);
	
    
    //matrixMulCUDA << < grid, threads >> >();
    //Comment out the above line and complete the shared memory version of the kernel
    matrixMulCUDASharedMemory << < grid, threads >> >();
    
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	checkCUDAError("CUDA kernel execution and timing");

	hipEventElapsedTime(&msec, start, stop);
	hipDeviceSynchronize();
	checkCUDAError("CUDA timing");

	// Ex 1.2.1: Compute the occupancy
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	occupancy = (deviceProp.maxBlocksPerMultiProcessor * threads.x * threads.y * threads.z) / (float)(deviceProp.maxThreadsPerMultiProcessor * deviceProp.multiProcessorCount);

	// Copy result from device to host
	hipMemcpyFromSymbol(h_C, HIP_SYMBOL(d_C), mem_size_C);
	checkCUDAError("CUDA memcpy results");

	// Compute reference CPU version
	matrixMulCPU(h_A, h_B, h_C_ref);

	// Check for errors
	errors = matrixMulTest(h_C, h_C_ref);
	if (errors)
		printf("%d total errors\n", errors);
	else
		printf("Test passed successfully\n");

	printf("Kernel time was %f with theoretical occupancy of %f\n", msec, occupancy);

}


void matrixMulCPU(float A[A_HEIGHT][A_WIDTH], float B[C_HEIGHT][C_WIDTH], float C[C_HEIGHT][C_WIDTH])
{
	int x, y, k;
	for (y = 0; y < C_HEIGHT; y++){
		for (x = 0; x < C_WIDTH; x++){
			C[y][x] = 0;
			for (k = 0; k < A_WIDTH; k++){
				C[y][x] += A[y][k] * B[k][x];
			}
		}
	}

}

int matrixMulTest(float C[C_HEIGHT][C_WIDTH], float Cref[C_HEIGHT][C_WIDTH])
{
	int errors = 0;
	int y, x;

	for (y = 0; y < C_HEIGHT; y++){
		for (x = 0; x < C_WIDTH; x++){
			if (C[y][x] != Cref[y][x]){
				errors++;
				printf("Device item c[%d][%d] = %f does not mach host result %f\n", y, x, C[y][x], Cref[y][x]);
			}
		}
	}

	return errors;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
