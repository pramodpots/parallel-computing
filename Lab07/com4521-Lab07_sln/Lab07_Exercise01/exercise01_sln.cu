#include <stdlib.h>
#include <stdio.h>
#include <math.h>


// include kernels and cuda headers after definitions of structures
#include "kernels.cuh" 


void checkCUDAError(const char*);
void readRecords(student_record *records);
// Ex 1.1 (2/3), It is necessary to declare the function's prototype, before it's used in main()
// Otherwise, the function would need to be implemented before main()
void studentRecordAOS2SOA(student_record *aos, student_records *soa);

void maximumMark_atomic(student_records*, student_records*, student_records*, student_records*);
void maximumMark_recursive(student_records*, student_records*, student_records*, student_records*);
void maximumMark_SM(student_records*, student_records*, student_records*, student_records*);
void maximumMark_shuffle(student_records*, student_records*, student_records*, student_records*);


int main(void) {
	student_record *recordsAOS;
	student_records *h_records;
	student_records *h_records_result;
	student_records *d_records;
	student_records *d_records_result;
	
	//host allocation
	recordsAOS = (student_record*)malloc(sizeof(student_record)*NUM_RECORDS);
	h_records = (student_records*)malloc(sizeof(student_records));
	h_records_result = (student_records*)malloc(sizeof(student_records));

	//device allocation
	hipMalloc((void**)&d_records, sizeof(student_records));
	hipMalloc((void**)&d_records_result, sizeof(student_records));
	checkCUDAError("CUDA malloc");

	//read file
	readRecords(recordsAOS);

	// Ex 1.1 (1/3), Convert recordsAOS to a structure of arrays in h_records
	studentRecordAOS2SOA(recordsAOS, h_records);
	
	//free AOS as it is no longer needed
	free(recordsAOS);

	//apply each approach in turn 
	maximumMark_atomic(h_records, h_records_result, d_records, d_records_result);
	maximumMark_recursive(h_records, h_records_result, d_records, d_records_result);
	maximumMark_SM(h_records, h_records_result, d_records, d_records_result);
	maximumMark_shuffle(h_records, h_records_result, d_records, d_records_result);


	// Cleanup
	free(h_records);
	free(h_records_result);
	hipFree(d_records);
	hipFree(d_records_result);
	checkCUDAError("CUDA cleanup");

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void readRecords(student_record *records){
	FILE *f = NULL;
	f = fopen("com4521.dat", "rb"); //read and binary flags
	if (f == NULL){
		fprintf(stderr, "Error: Could not find com4521.dat file \n");
		exit(1);
	}

	//read student data
	if (fread(records, sizeof(student_record), NUM_RECORDS, f) != NUM_RECORDS){
		fprintf(stderr, "Error: Unexpected end of file!\n");
		exit(1);
	}
	fclose(f);
}

// Ex 1.1 (3/3), This function simplies iterates the structures, and copies the data from aos to soa
void studentRecordAOS2SOA(student_record *aos, student_records *soa){
	for (int i = 0; i < NUM_RECORDS; i++){
		soa->student_ids[i] = aos[i].student_id;
		soa->assignment_marks[i] = aos[i].assignment_mark;
	}
}


void maximumMark_atomic(student_records *h_records, student_records *h_records_result, student_records *d_records, student_records *d_records_result){
	float max_mark;
	int max_mark_student_id;
	float time;
	hipEvent_t start, stop;

	max_mark = 0.0f;
	max_mark_student_id = 0;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	//memory copy records to device
	hipMemcpy(d_records, h_records, sizeof(student_records), hipMemcpyHostToDevice);
	checkCUDAError("1) CUDA memcpy");

	hipEventRecord(start, 0);
	//find highest mark using GPU
	dim3 blocksPerGrid(NUM_RECORDS / THREADS_PER_BLOCK, 1, 1);
	dim3 threadsPerBlock(THREADS_PER_BLOCK, 1, 1);
	maximumMark_atomic_kernel << <blocksPerGrid, threadsPerBlock >> >(d_records);
	hipDeviceSynchronize();
	checkCUDAError("Atomics: CUDA kernel");

	// Copy result back to host
	hipMemcpyFromSymbol(&max_mark, HIP_SYMBOL(d_max_mark), sizeof(float));
	hipMemcpyFromSymbol(&max_mark_student_id, HIP_SYMBOL(d_max_mark_student_id), sizeof(int));
	checkCUDAError("Atomics: CUDA memcpy back");

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	//output result
	printf("Atomics: Highest mark recorded %f was by student %d\n", max_mark, max_mark_student_id);
	printf("\tExecution time was %f ms\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}

// Exercise 2)
void maximumMark_recursive(student_records *h_records, student_records *h_records_result, student_records *d_records, student_records *d_records_result){
	int i;
	float max_mark;
	int max_mark_student_id;
	student_records *d_records_temp;
	float time;
	hipEvent_t start, stop;

	max_mark = 0.0f;
	max_mark_student_id = 0;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// memory copy records to device
	hipMemcpy(d_records, h_records, sizeof(student_records), hipMemcpyHostToDevice);
	checkCUDAError("Recursive: CUDA memcpy");

	hipEventRecord(start, 0);
	
	// Ex 2.3, Iteratively call GPU steps so that there are THREADS_PER_BLOCK values left
	for (i = NUM_RECORDS; i > THREADS_PER_BLOCK; i /= 2){

		dim3 blocksPerGrid(i / THREADS_PER_BLOCK, 1, 1);
		dim3 threadsPerBlock(THREADS_PER_BLOCK, 1, 1);
		maximumMark_recursive_kernel << <blocksPerGrid, threadsPerBlock, sizeof(student_record)*THREADS_PER_BLOCK >> >(d_records, d_records_result);
		hipDeviceSynchronize();
		checkCUDAError("Recursive: CUDA kernel");

		//swap input and output
		d_records_temp = d_records;
		d_records = d_records_result;
		d_records_result = d_records_temp;
	}

	// Ex 2.4, copy back the final THREADS_PER_BLOCK values
	hipMemcpy(h_records_result->assignment_marks, d_records->assignment_marks, sizeof(float)*THREADS_PER_BLOCK, hipMemcpyDeviceToHost);
	hipMemcpy(h_records_result->student_ids, d_records->student_ids, sizeof(int)*THREADS_PER_BLOCK, hipMemcpyDeviceToHost);
	checkCUDAError("Recursive: CUDA memcpy back");

	// Ex 2.5, reduce the final THREADS_PER_BLOCK values on CPU
	max_mark = 0;
	max_mark_student_id = 0;
	for (i = 0; i < THREADS_PER_BLOCK; i++){
		float mark = h_records_result->assignment_marks[i];
		if (mark > max_mark){
			max_mark = mark;
			max_mark_student_id = h_records_result->student_ids[i];;
		}
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	// output the result
	printf("Recursive: Highest mark recorded %f was by student %d\n", max_mark, max_mark_student_id);
	printf("\tExecution time was %f ms\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}

// Ex 3)
void maximumMark_SM(student_records *h_records, student_records *h_records_result, student_records *d_records, student_records *d_records_result){
	unsigned int i;
	float max_mark;
	int max_mark_student_id;
	float time;
	hipEvent_t start, stop;

	max_mark = 0.0f;
	max_mark_student_id = 0;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	//memory copy records to device
	hipMemcpy(d_records, h_records, sizeof(student_records), hipMemcpyHostToDevice);
	checkCUDAError("SM: CUDA memcpy");

	// Ex 3.4, Call the shared memory reduction kernel
	hipEventRecord(start, 0);
	dim3 blocksPerGrid(NUM_RECORDS / THREADS_PER_BLOCK, 1, 1);
	dim3 threadsPerBlock(THREADS_PER_BLOCK, 1, 1);
	maximumMark_SM_kernel << <blocksPerGrid, threadsPerBlock, sizeof(student_record)*THREADS_PER_BLOCK >> >(d_records, d_records_result);
	hipDeviceSynchronize();
	checkCUDAError("SM: CUDA kernel");


	// Ex 3.5, Copy the final block values back to CPU
	hipMemcpy(h_records_result->assignment_marks, d_records_result->assignment_marks, sizeof(float)*blocksPerGrid.x, hipMemcpyDeviceToHost);
	hipMemcpy(h_records_result->student_ids, d_records_result->student_ids, sizeof(int)*blocksPerGrid.x, hipMemcpyDeviceToHost);
	checkCUDAError("SM: CUDA memcpy back");
	max_mark = 0;
	max_mark_student_id = 0;

	// Ex 3.6, Reduce the block level results on CPU
	for (i = 0; i < blocksPerGrid.x; i++){
		float mark = h_records_result->assignment_marks[i];
		if (mark > max_mark){
			max_mark = mark;
			max_mark_student_id = h_records_result->student_ids[i];;
		}
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	// output result
	printf("SM: Highest mark recorded %f was by student %d\n", max_mark, max_mark_student_id);
	printf("\tExecution time was %f ms\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}

// Exercise 4)
void maximumMark_shuffle(student_records *h_records, student_records *h_records_result, student_records *d_records, student_records *d_records_result){
	unsigned int i;
	unsigned int warps_per_grid;
	float max_mark;
	int max_mark_student_id;
	float time;
	hipEvent_t start, stop;

	max_mark = 0.0f;
	max_mark_student_id = 0;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// memory copy records to device
	hipMemcpy(d_records, h_records, sizeof(student_records), hipMemcpyHostToDevice);
	checkCUDAError("Shuffle: CUDA memcpy");

	// Ex 4.2, Execute the kernel
	hipEventRecord(start, 0);
	dim3 blocksPerGrid(NUM_RECORDS / THREADS_PER_BLOCK, 1, 1);
	dim3 threadsPerBlock(THREADS_PER_BLOCK, 1, 1);
	warps_per_grid = NUM_RECORDS / 32;
	maximumMark_shuffle_kernel << <blocksPerGrid, threadsPerBlock >> >(d_records, d_records_result);
	hipDeviceSynchronize();
	checkCUDAError("Shuffle: CUDA kernel");

	// copy the final warp values back to CPU
	hipMemcpy(h_records_result->assignment_marks, d_records_result->assignment_marks, sizeof(float)*warps_per_grid, hipMemcpyDeviceToHost);
	hipMemcpy(h_records_result->student_ids, d_records_result->student_ids, sizeof(int)*warps_per_grid, hipMemcpyDeviceToHost);
	checkCUDAError("Shuffle: CUDA memcpy back");
	max_mark = 0;
	max_mark_student_id = 0;

	// reduce the warp level results on CPU
	for (i = 0; i < warps_per_grid; i++){
		float mark = h_records_result->assignment_marks[i];
		if (mark > max_mark){
			max_mark = mark;
			max_mark_student_id = h_records_result->student_ids[i];;
		}
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	// output result
	printf("Shuffle: Highest mark recorded %f was by student %d\n", max_mark, max_mark_student_id);
	printf("\tExecution time was %f ms\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}