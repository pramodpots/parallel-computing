#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

//The number of character in the encrypted text
#define N 1024

void checkCUDAError(const char*);
void read_encrypted_file(int*);

#define A 15
#define B 27
#define M 128
#define A_MMI_M 111

// Ex 1.1, Device functions are preceded by __device__
// These can only be called from other device functions or kernels (__global__)
__device__ int modulo(int a, int b){
	int r = a % b;
	r = (r < 0) ? r + b : r;
	return r;
}

// Ex 1.2, threadIdx provides access to the thread's index within the block
// Thread blocks may be 3 dimensional threadIdx.x, threadIdx.y, threadIdx.z
// However in this case, the y and z widths should both equal 1, so those indexes will return 0
__global__ void affine_decrypt(int *d_input, int *d_output)
{
	int index = threadIdx.x;
	int value = d_input[index];
	value = modulo(A_MMI_M * (value - B), M);
	d_output[index] = value;
}

// Ex 1.8 (1/2), As we are using multiple blocks, it's necessary to consider the block index when calculating a global thread index
// blockDim provides the dimensions of a block
// blockIdx provides the current thread's block index, similar to how threadIdx works
// threadIdx provides the threads index, relative to it's block!
__global__ void affine_decrypt_multiblock(int *d_input, int *d_output)
{
	int index = blockDim.x*blockIdx.x + threadIdx.x;
	int value = d_input[index];
	value = modulo(A_MMI_M * (value - B), M);
	d_output[index] = value;
}


int main(int argc, char *argv[])
{
	int *h_input, *h_output;
	int *d_input, *d_output;
	unsigned int size;
	int i;

	size = N * sizeof(int);

	/* allocate the host memory */
	h_input = (int *)malloc(size);
	h_output = (int *)malloc(size);

	// Ex 1.3, hipMalloc() does not return the pointer to memory, it stores it in the memory location you provide
	// In this way, it differs from the malloc() you use for host memory
	// hipMalloc() returns an error code
    /* allocate device memory */
	hipMalloc((void **)&d_input, size);
	hipMalloc((void **)&d_output, size);
	checkCUDAError("Memory allocation");

	/* read the encryted text */
	read_encrypted_file(h_input);

	// Ex 1.4, hipMemcpy() works similar to memcpy(), however an additional argument specifying the type of copy must be provided
	// hipMemcpy() returns an error code
	/* copy host input to device input */
	hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
	checkCUDAError("Input transfer to device");

	/* Configure the grid of thread blocks and run the GPU kernel */
	// Ex 1.5, grid and block sizes are specified within the triple chevrons (<<< >>>)
	// affine_decrypt<<<1, N>>> (d_input, d_output);

	// Ex 1.8 (2/2), dim3 objects can be used to specify the grid and block dimensions
	// However, these are only strictly necessary for 2D and 3D launches
	dim3 blocksPerGrid(8, 1, 1);
	dim3 threadsPerBlock(N / 8, 1, 1);
	affine_decrypt_multiblock<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output);

	/* wait for all threads to complete */
	hipDeviceSynchronize();
	checkCUDAError("Kernel execution");

	// Ex 1.6, The additional argument is changed, in order to copy data back from the device
	/* copy the gpu output back to the host */
	hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);
	checkCUDAError("Result transfer to host");

	/* print out the result to screen */
	for (i = 0; i < N; i++) {
		printf("%c", (char)h_output[i]);
	}
	printf("\n");

	/* free device memory */
	hipFree(d_input);
	hipFree(d_output);
	checkCUDAError("Free memory");

	/* free host buffers */
	free(h_input);
	free(h_output);

	return 0;
}


void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void read_encrypted_file(int* input)
{
	FILE *f = NULL;
	f = fopen("encrypted01.bin", "rb"); //read and binary flags
	if (f == NULL){
		fprintf(stderr, "Error: Could not find encrypted01.bin file \n");
		exit(1);
	}
	//read encrypted data
	fread(input, sizeof(unsigned int), N, f);
	fclose(f);
}