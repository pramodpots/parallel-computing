#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"


// Ex 3.1, Set N to 2048
#define N 2048 //width
// Ex 3.5 Define height separately
#define M 1000 //height
#define THREADS_PER_BLOCK 256
#define SQRT_THREADS_PER_BLOCK sqrt(THREADS_PER_BLOCK)

void checkCUDAError(const char*);
void random_ints(int *a);
void matrixAddCPU(int *a, int *b, int *c);
int validate(int *a, int *ref);


__global__ void matrixAdd(int *a, int *b, int *c) {
	// Ex 3.4 (2/2), Calculate a global 2D index for the thread
	// Then later on, convert this to a 1D index in i
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	//check bounds
	if ((x < N) && (y < M)){
		int i = y*N + x;
		c[i] = a[i] + b[i];
	}
}



int main(void) {
	int *a, *b, *c, *c_ref;			// host copies of a, b, c
	int *d_a, *d_b, *d_c;			// device copies of a, b, c
	int errors;
	// Ex 3.1/3.5, Modify size to account for 2 dimensions
	unsigned int size = N * M * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	checkCUDAError("CUDA malloc");

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a);
	b = (int *)malloc(size); random_ints(b);
	c = (int *)malloc(size);
	c_ref = (int *)malloc(size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy");

	// Ex 3.4 (1/2), Specify the y dimensions for the blocks and grid
	// Launch add() kernel on GPU
	unsigned int block_width = (unsigned int)SQRT_THREADS_PER_BLOCK;
	unsigned int grid_width = (unsigned int)ceil((double)N / block_width);
	unsigned int grid_height = (unsigned int)ceil((double)M / block_width);
	dim3 blocksPerGrid(grid_width, grid_height, 1);
	dim3 threadsPerBlock(block_width, block_width, 1);
	matrixAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);
	checkCUDAError("CUDA kernel");

	//perform CPU version
	matrixAddCPU(a, b, c_ref);


	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	checkCUDAError("CUDA memcpy");

	//validate
	errors = validate(c, c_ref);
	printf("CUDA GPU result has %d errors.\n", errors);



	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	checkCUDAError("CUDA cleanup");

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

// Ex 3.2, Modify the function to fill the full matrix
void random_ints(int *a)
{
	for (unsigned int x = 0; x < N; x++){
		for (unsigned int y = 0; y < M; y++){
			a[(y*N) + x] = rand();
		}
	}
}

// Ex 3.3 (1/2), Modify the function to handle the full matrix
void matrixAddCPU(int *a, int *b, int *c)
{
	for (unsigned int x = 0; x < N; x++){
		for (unsigned int y = 0; y < M; y++){
			int i = (y*N) + x;
			c[i] = a[i] + b[i];
		}
	}
}

// Ex 3.3 (2/2), Modify the function to handle the full matrix
int validate(int *a, int *ref){
	int errors = 0;
	for (unsigned int x = 0; x < N; x++){
		for (unsigned int y = 0; y < M; y++){
			int i = (y*N) + x;
			if (a[i] != ref[i]){
				errors++;
				fprintf(stderr, "ERROR at index %d: GPU result %d does not match CPU value of %d\n", i, a[i], ref[i]);
			}
		}
	}

	return errors;
}